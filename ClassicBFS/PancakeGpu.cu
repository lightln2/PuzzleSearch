#include "hip/hip_runtime.h"
#include "SlidingPuzzleGpu.h"

#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <stdio.h>

constexpr uint64_t INVALID_INDEX = std::numeric_limits<uint64_t>::max();

namespace {

    __device__ bool HasOp(int op, int dir) {
        return op & (1 << dir);
    }

    __device__ void gpu_swap(int& x, int& y) {
        int temp = x;
        x = y;
        y = temp;
    }

    __device__ void MoveInternal(int* newarr, int size, int count) {
        int start = size - count, end = size - 1;
        while (start < end) {
            gpu_swap(newarr[start], newarr[end]);
            start++;
            end--;
        }
    }

    __device__ void Move(int* arr, int* newarr, int size, int count) {
        for (int i = 0; i < 16; i++) newarr[i] = arr[i];
        MoveInternal(newarr, size, count);
    }

    __device__ uint64_t OptPermutationRank(int* arr, int size) {
        GpuPermutationCompact(arr, size);
        if (size <= 12) {
            uint64_t index = 0;
            for (int i = 0; i < size; i++) {
                index *= (size - i);
                index += arr[i];
            }

            return index;
        }
        else {
            uint64_t index = 0;
            for (int i = size - 12; i < size; i++) {
                index *= (size - i);
                index += arr[i];
            }

            uint64_t segment = 0;
            for (int i = 0; i < size - 12; i++) {
                segment *= (size - i);
                segment += arr[i];
            }

            return (segment << 29) | index;
        }
    }

    __device__ void OptPermutationUnrank(uint64_t index, int* arr, int size) {
        if (size <= 12) {
            for (int i = size - 1; i >= 0; i--) {
                arr[i] = index % (size - i);
                index /= (size - i);
            }
        }
        else {
            uint64_t segment = index >> 29;
            index &= ((1ui64 << 29) - 1);

            for (int i = size - 13; i >= 0; i--) {
                arr[i] = segment % (size - i);
                segment /= (size - i);
            }

            for (int i = size - 1; i >= size - 12; i--) {
                arr[i] = index % (size - i);
                index /= (size - i);
            }
        }

        GpuPermutationUncompact(arr, size);
    }

}

__global__ void kernel_pancake_expand(uint64_t* indexes, uint64_t* expanded, int size, uint64_t count) {
    uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= count) return;

    const int MAX_OP = size - 1;
    const uint64_t OP_MASK = (1ui64 << MAX_OP) - 1;

    uint64_t index = indexes[i];
    int opBits = index & OP_MASK;
    index >>= MAX_OP;

    int arr[16];
    OptPermutationUnrank(index, arr, size);

    int newarr[16];

    uint64_t dstBase = i * MAX_OP;
    for (int op = 0; op < MAX_OP; op++) {
        uint64_t result = INVALID_INDEX;
        if (!HasOp(opBits, op)) {
            Move(arr, newarr, size, op + 2);
            uint64_t child = OptPermutationRank(newarr, size);
            result = (child << MAX_OP) | op;
        }
        expanded[dstBase + op] = result;
    }
}

__global__ void kernel_pancake_expandInSegment(uint64_t* indexes, uint64_t* expanded, int size, uint64_t count) {
    uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= count) return;

    const int MAX_OP = size - 1;
    const uint64_t OP_MASK = (1ui64 << MAX_OP) - 1;

    uint64_t index = indexes[i];
    int opBits = index & OP_MASK;
    index >>= MAX_OP;

    int arr[16];
    OptPermutationUnrank(index, arr, size);

    int newarr[16];

    uint64_t dstBase = i * 11;
    for (int op = 0; op < 11; op++) {
        uint64_t result = INVALID_INDEX;
        if (!HasOp(opBits, op)) {
            Move(arr, newarr, size, op + 2);
            uint64_t child = OptPermutationRank(newarr, size);
            result = (child << MAX_OP) | op;
        }
        expanded[dstBase + op] = result;
    }
}

__global__ void kernel_pancake_expandCrossSegment(uint64_t* indexes, uint64_t* expanded, int size, bool invIndex, uint64_t count) {
    uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= count) return;

    const int MAX_OP = size - 1;
    const uint64_t OP_MASK = (1ui64 << MAX_OP) - 1;

    uint64_t index = indexes[i];
    int opBits = index & OP_MASK;
    index >>= MAX_OP;

    int arr[16];
    OptPermutationUnrank(index, arr, size);

    int newarr[16];

    uint64_t dstBase = i * (MAX_OP - 11) - 11;
    for (int op = 11; op < MAX_OP; op++) {
        uint64_t result = INVALID_INDEX;
        if (!HasOp(opBits, op)) {
            Move(arr, newarr, size, op + 2);
            if (size > 12 && invIndex) {
                MoveInternal(newarr, size, 12);
            }
            uint64_t child = OptPermutationRank(newarr, size);
            result = (child << MAX_OP) | op;
        }
        expanded[dstBase + op] = result;
    }
}

__global__ void kernel_pancake_crossSegmentPostProcess(uint32_t* indexes, int segment, int size, uint64_t count) {
    uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= count) return;

    const uint64_t SEG_MASK = (1ui64 << 29) - 1;

    uint64_t index = (uint64_t(segment) << 29) | indexes[i];

    int arr[16];
    OptPermutationUnrank(index, arr, size);
    MoveInternal(arr, size, 12);
    index = OptPermutationRank(arr, size);
    if (segment != index >> 29) index = INVALID_INDEX;
    indexes[i] = uint32_t(index & SEG_MASK);
}

void PancakeExpand(
    uint64_t* gpuIndexes,
    uint64_t* gpuExpanded,
    int size,
    uint64_t count,
    CuStream stream)
{
    auto threadsPerBlock = 256;
    auto blocksPerGrid = uint32_t((count + threadsPerBlock - 1) / threadsPerBlock);
    
    kernel_pancake_expand<<<blocksPerGrid, threadsPerBlock, 0, hipStream_t(stream) >>> (
        gpuIndexes,
        gpuExpanded,
        size,
        count);
    ERR(hipGetLastError());
}

void PancakeExpandInSegment(
    uint64_t* gpuIndexes,
    uint64_t* gpuExpanded,
    int size,
    uint64_t count,
    CuStream stream)
{
    auto threadsPerBlock = 256;
    auto blocksPerGrid = uint32_t((count + threadsPerBlock - 1) / threadsPerBlock);

    kernel_pancake_expandInSegment << <blocksPerGrid, threadsPerBlock, 0, hipStream_t(stream) >> > (
        gpuIndexes,
        gpuExpanded,
        size,
        count);
    ERR(hipGetLastError());
}

void PancakeExpandCrossSegment(
    uint64_t* gpuIndexes,
    uint64_t* gpuExpanded,
    int size,
    bool invIndex,
    uint64_t count,
    CuStream stream)
{
    auto threadsPerBlock = 256;
    auto blocksPerGrid = uint32_t((count + threadsPerBlock - 1) / threadsPerBlock);

    kernel_pancake_expandCrossSegment << <blocksPerGrid, threadsPerBlock, 0, hipStream_t(stream) >> > (
        gpuIndexes,
        gpuExpanded,
        size,
        invIndex,
        count);
    ERR(hipGetLastError());
}

void PancakeCrossSegmentPostProcessGPU(
    uint32_t* gpuIndexes,
    int segment,
    int size,
    uint64_t count,
    CuStream stream)
{
    auto threadsPerBlock = 256;
    auto blocksPerGrid = uint32_t((count + threadsPerBlock - 1) / threadsPerBlock);

    kernel_pancake_crossSegmentPostProcess << <blocksPerGrid, threadsPerBlock, 0, hipStream_t(stream) >> > (
        gpuIndexes,
        segment,
        size,
        count);
    ERR(hipGetLastError());
}
