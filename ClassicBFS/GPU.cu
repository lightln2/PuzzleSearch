#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <iostream>

static void ERR(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << hipGetErrorString(err) << std::endl;
        throw std::runtime_error(hipGetErrorString(err));
    }
}

uint64_t* CreateGPUBuffer(int count) {
    uint64_t* gpuBuffer;
    ERR(hipMalloc((void**)&gpuBuffer, count * sizeof(int64_t)));
    return gpuBuffer;
}

void DestroyGPUBuffer(uint64_t* gpuBuffer) {
    ERR(hipFree(gpuBuffer));
}

void CopyToGpu(uint64_t* buffer, uint64_t* gpuBuffer, size_t count) {
    ERR(hipMemcpy(gpuBuffer, buffer, count * sizeof(uint64_t), hipMemcpyHostToDevice));
}

void CopyFromGpu(uint64_t* gpuBuffer, uint64_t* buffer, size_t count) {
    ERR(hipMemcpy(buffer, gpuBuffer, count * sizeof(int64_t), hipMemcpyDeviceToHost));
}


__device__ void GpuPermutationCompact(int* arr, int size) {
    int set_bits = 0;

    auto cntBits = [&](int index) {
        return __popcll(set_bits & ((1 << index) - 1));
    };

    for (int i = 0; i < size; i++) {
        int tile = arr[i];
        arr[i] -= cntBits(tile);
        set_bits |= (1 << tile);
    }
}

__device__ void GpuPermutationUncompact(int* arr, int size) {
    uint64_t tiles = 0xFEDCBA9876543210ui64;
    auto gettile = [&](int index) {
        return (int)(tiles >> (index * 4)) & 15;
    };
    auto removetile = [&](int index) {
        auto hi_tiles = (tiles >> (index * 4 + 4)) << (index * 4);
        auto lo_tiles = tiles & ((1ui64 << (index * 4)) - 1);
        tiles = hi_tiles | lo_tiles;
    };

    for (int i = 0; i < size; i++) {
        int tile = arr[i];
        arr[i] = gettile(tile);
        removetile(tile);
    }
}

__device__ uint64_t GpuPermutationRank(int* arr, int size) {
    GpuPermutationCompact(arr, size);

    uint64_t index = 0;
    for (int i = 0; i < size; i++) {
        index *= (size - i);
        index += arr[i];
    }

    return index;
}

__device__ void GpuPermutationUnrank(uint64_t index, int* arr, int size) {
    for (int i = size - 1; i >= 0; i--) {
        arr[i] = index % (size - i);
        index /= (size - i);
    }

    GpuPermutationUncompact(arr, size);
}


/* *** SLIDING TILE PUZLE *** */

constexpr int OP_UP = 0, OP_LEFT = 1, OP_RIGHT = 2, OP_DOWN = 3;
constexpr uint64_t INVALID_INDEX = std::numeric_limits<uint64_t>::max();

__device__ bool HasOp(int op, int dir) {
    return op & (1 << dir);
}

__device__ void Move(int* arr, int* newarr, int blank, int newblank) {
    for (int i = 0; i < 16; i++) newarr[i] = arr[i];
    newarr[blank] = arr[newblank];
    newarr[newblank] = arr[blank];
}

__global__ void kernel_sliding_tile_simple(uint64_t* indexes, uint64_t* expanded, int width, int size, uint64_t count) {
    uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= count) return;

    uint64_t index = indexes[i];
    int opBits = index & 15;
    index >>= 4;

    int arr[16];
    GpuPermutationUnrank(index, arr, size);

    int blank = -1;
    for (int i = 0; i < size; i++) {
        if (arr[i] == 0) {
            blank = i;
            //break;
        }
    }

    int newarr[16];

    if (blank >= width && !HasOp(opBits, OP_UP)) {
        Move(arr, newarr, blank, blank - width);
        uint64_t exp = GpuPermutationRank(newarr, size);
        expanded[i * 4 + 0] = (exp << 4) | OP_DOWN;
    }
    else {
        expanded[i * 4 + 0] = INVALID_INDEX;
    }

    if (blank < size - width && !HasOp(opBits, OP_DOWN)) {
        Move(arr, newarr, blank, blank + width);
        uint64_t exp = GpuPermutationRank(newarr, size);
        expanded[i * 4 + 1] = (exp << 4) | OP_UP;
    }
    else {
        expanded[i * 4 + 1] = INVALID_INDEX;
    }

    if (blank % width > 0 && !HasOp(opBits, OP_LEFT)) {
        Move(arr, newarr, blank, blank - 1);
        uint64_t exp = GpuPermutationRank(newarr, size);
        expanded[i * 4 + 2] = (exp << 4) | OP_RIGHT;
    }
    else {
        expanded[i * 4 + 2] = INVALID_INDEX;
    }

    if (blank % width < width - 1 && !HasOp(opBits, OP_RIGHT)) {
        Move(arr, newarr, blank, blank + 1);
        uint64_t exp = GpuPermutationRank(newarr, size);
        expanded[i * 4 + 3] = (exp << 4) | OP_LEFT;
    }
    else {
        expanded[i * 4 + 3] = INVALID_INDEX;
    }
}

void GpuSlidingTilePuzzleSimpleExpand(
    uint64_t* gpuIndexes,
    uint64_t* gpuExpanded,
    int width,
    int size,
    uint64_t count)
{
    auto threadsPerBlock = 256;
    auto blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;
    kernel_sliding_tile_simple<<<blocksPerGrid, threadsPerBlock >>> (gpuIndexes, gpuExpanded, width, size, count);
    ERR(hipGetLastError());
}
